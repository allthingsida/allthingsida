#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <thread>

#define PASSWORD_LENGTH 8
#define CHARSET_SIZE 26
#define MAX_SOLUTIONS 1024 // Maximum number of solutions buffer size

// Constants and charset for the kernel
__device__ const char charset[CHARSET_SIZE + 1] = "abcdefghijklmnopqrstuvwxyz";
__device__ const uint32_t MOD_ADLER = 65521;  // Largest prime < 2^16

// Compute Adler-32 checksum
__device__ uint32_t adler32(const char *data, size_t len) {
    uint32_t a = 1, b = 0;
    for (size_t i = 0; i < len; i++) {
        a = (a + data[i]) % MOD_ADLER;
        b = (b + a) % MOD_ADLER;
    }
    return (b << 16) | a;
}

// CUDA kernel to check each possible password
__global__ void find_passwords(uint32_t target, char *results, int *flags) {
    // Calculate global index for each thread
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Generate an 8-character password based on the thread's index
    char candidate[PASSWORD_LENGTH + 1];
    uint64_t temp_idx = idx;

    // Decode the index into a password
    for (int i = 0; i < PASSWORD_LENGTH; i++) {
        candidate[i] = charset[temp_idx % CHARSET_SIZE];
        temp_idx /= CHARSET_SIZE;
    }
    candidate[PASSWORD_LENGTH] = '\0';  // Null-terminate the string

    // Check the Adler-32 checksum
    if (adler32(candidate, PASSWORD_LENGTH) == target) {
        // Find an available slot
        for (int pos = 0; pos < MAX_SOLUTIONS; pos++) {
            if (atomicCAS(&flags[pos], 0, 1) == 0) {  // Reserve slot
                // Store the result
                for (int i = 0; i < PASSWORD_LENGTH; i++) {
                    results[pos * PASSWORD_LENGTH + i] = candidate[i];
                }
                atomicExch(&flags[pos], 2);  // Mark as filled
                break;
            }
        }
    }
}

// Host function to launch the CUDA kernel and poll for results
void launch_brute_force(uint32_t target) {
    // Allocate memory for results and flags on the device
    char *d_results;
    int *d_flags;
    hipMalloc((void**)&d_results, MAX_SOLUTIONS * PASSWORD_LENGTH * sizeof(char));
    hipMalloc((void**)&d_flags, MAX_SOLUTIONS * sizeof(int));
    hipMemset(d_flags, 0, MAX_SOLUTIONS * sizeof(int));

    // Kernel launch configuration
    uint64_t total_combinations = pow(CHARSET_SIZE, PASSWORD_LENGTH);
    int threads_per_block = 256;
    int blocks = (total_combinations + threads_per_block - 1) / threads_per_block;

    // Launch the kernel asynchronously
    find_passwords<<<blocks, threads_per_block>>>(target, d_results, d_flags);

    // Host polling loop to check for results
    char h_results[PASSWORD_LENGTH + 1];
    h_results[PASSWORD_LENGTH] = '\0';  // Null-terminate for safe printing

    printf("Searching for passwords matching the checksum...\n");
    while (true) {
        bool any_found = false;

        // Check each slot in the flags array
        for (int i = 0; i < MAX_SOLUTIONS; i++) {
            int flag;
            hipMemcpy(&flag, &d_flags[i], sizeof(int), hipMemcpyDeviceToHost);

            if (flag == 2) {  // Result available
                // Copy the result from device to host
                hipMemcpy(h_results, &d_results[i * PASSWORD_LENGTH], PASSWORD_LENGTH * sizeof(char), hipMemcpyDeviceToHost);

                // Print the password
                printf("found: %s\n", h_results);
                
                // Mark slot as empty again for reuse
                int zero = 0;
                hipMemcpy(&d_flags[i], &zero, sizeof(int), hipMemcpyHostToDevice);
                
                any_found = true;
            }
        }

        if (!any_found) {
            // Sleep for a short time to avoid busy waiting
            std::this_thread::sleep_for(std::chrono::milliseconds(100));
        }
    }

    // Free device memory
    hipFree(d_results);
    hipFree(d_flags);
}

#define MAGIC3 0x0F910374  // Target Adler-32 checksum

int main() {
    // Call the brute-force function with the target Adler-32 checksum
    launch_brute_force(MAGIC3);
    return 0;
}
